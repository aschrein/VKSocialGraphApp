#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <CudaModule/CudaModule.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <unordered_set>
struct TransientQuadNode
{
	vec2 position;
	float size;
	__host__ __device__ bool contains( vec2 const &point )
	{
		return fabsf( position.x - point.x ) <= size &&  fabsf( position.y - point.y ) <= size;
	}
};
struct LeafMapping
{
	int m_leafId;
	int m_index;
};
#define cuAssert( x ) if( !(x) ){ return; }
/*
 routine used to map points to nodes and count total occupation per node
*/
__global__ void distributeCells(
	vec2 const *pPoints , LeafMapping *dPointsToLeafs , int posN ,
	float centerX , float centerY , float cellSize , QuadNode *pNodes )
{
	int pointId = threadIdx.x + blockDim.x * blockIdx.x;

	cuAssert( pointId < posN );

	vec2 point = pPoints[ pointId ];
	QuadNode curNode = pNodes[ 0 ];
	int curIndex = 0 , oldIndex = 0;
	//the average depth is log4(N)
	while( curNode.children[ 0 ] > 0 )
	{
		for( int i = 0; i < 4; i++ )
		{
			float childCenterX = centerX + cellSize * ( ( i & 1 ) * 2 - 1 ) / 2;
			float childCenterY = centerY + cellSize * ( ( i >> 1 ) * 2 - 1 ) / 2;
			float childSize = cellSize / 2;
			if(
				TransientQuadNode{ { childCenterX , childCenterY} , childSize }
				.contains( point )
				)
			{
				//no other node contains this point and we must switch to that node and check against its children
				curIndex = curNode.children[ i ];
				curNode = pNodes[ curIndex ];
				centerX = childCenterX;
				centerY = childCenterY;
				cellSize = childSize;
				break;
			}
		}
		cuAssert( curIndex != oldIndex );
		oldIndex = curIndex;
	}
	//increase the node's occupation counter
	int index = atomicAdd( &pNodes[ curIndex ].itemsCount , 1 );
	//map this point to the leaf
	dPointsToLeafs[ pointId ] = { curIndex , index };
}
__global__ void mapOrder( QuadNode *pNodes , int leafsCount , int *pLeafIndices , int *pCounts )
{
	int leafIndex = threadIdx.x + blockDim.x * blockIdx.x;
	cuAssert( leafIndex < leafsCount );
	pNodes[ pLeafIndices[ leafIndex ] ].order = leafIndex;
	pCounts[ leafIndex ] = pNodes[ pLeafIndices[ leafIndex ] ].itemsCount;
}
__global__ void fillLeafs( QuadNode *pNodes , int leafsCount , int *pLeafIndices , int const *pCountsScan )
{
	int leafIndex = threadIdx.x + blockDim.x * blockIdx.x;
	cuAssert( leafIndex < leafsCount );
	pNodes[ pLeafIndices[ leafIndex ] ].itemsBegin = -pCountsScan[ leafIndex ];
}
__global__ void kPack( float const *pos , int posN , float *npos )
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if( i < posN )
	{

	}
}
//device side array
template< class T > using dVector = thrust::device_vector< T >;
//host side array
template< class T > using hVector = std::vector< T >;
template< class T > using hSet = std::unordered_set< T >;
/*
allocates 4 more nodes and associate ith node to them as parent
puts indices of the children to the "aIndices"
*/
void splitLeaf(
	hVector< QuadNode > &hQuadNodes ,
	int leafIndex ,
	hSet< int > &aLeafsIndices
)
{
	int count = hQuadNodes.size();
	hQuadNodes[ leafIndex ] = QuadNode{ count , count + 1 , count + 2 , count + 3 };
	hQuadNodes.push_back( { -1 , 0 , 0 , 0 } );
	hQuadNodes.push_back( { -1 , 0 , 0 , 0 } );
	hQuadNodes.push_back( { -1 , 0 , 0 , 0 } );
	hQuadNodes.push_back( { -1 , 0 , 0 , 0 } );
	aLeafsIndices.erase( leafIndex );
	aLeafsIndices.insert( count );
	aLeafsIndices.insert( count + 1 );
	aLeafsIndices.insert( count + 2 );
	aLeafsIndices.insert( count + 3 );
}
template< typename T >
void copy( hVector< T > &hVecDst , dVector< T > const &dVecSrc )
{
	hVecDst.resize( dVecSrc.size() );
	thrust::copy( dVecSrc.begin() , dVecSrc.end() , hVecDst.begin() );
}
template< typename T >
void copy( dVector< T > &dVecDst , hVector< T > const &hVecSrc )
{
	dVecDst.resize( hVecSrc.size() );
	thrust::copy( hVecSrc.begin() , hVecSrc.end() , dVecDst.begin() );
}
void packCuda( hVector< Relation > const &relations ,
	hVector< vec2 > &aPoints ,
	hVector< QuadNode > &out_aQuadNode )
{
	//calculate the point system's spatial extents
	float max_x = 0.0f , min_x = 0.0f , max_y = 0.0f , min_y = 0.0f;
	for( auto const &pos : aPoints )
	{
		max_x = fmaxf( max_x , pos.x );
		min_x = fminf( min_x , pos.x );
		max_y = fmaxf( max_y , pos.y );
		min_y = fminf( min_y , pos.y );
	}
	float rootX = ( max_x + min_x ) * 0.5f;
	float rootY = ( max_y + min_y ) * 0.5f;
	float rootSize = fmaxf( ( max_x - min_x ) * 0.5f , ( max_y - min_y ) * 0.5f );

	//device side copy of points
	dVector< vec2 > dPoints = aPoints;
	//points to nodes mapping
	dVector< LeafMapping > dPointsToLeafs( aPoints.size() );
	//array of quad nodes
	dVector< QuadNode > dQuadNodes;
	hVector< QuadNode > hQuadNodes;
	//indices of leafs in dQuadNodes
	hSet< int > aLeafsIndices;
	//push root node
	hQuadNodes.push_back( { } );
	//allocate 4 leafs
	splitLeaf( hQuadNodes , 0 , aLeafsIndices );
	copy( dQuadNodes , hQuadNodes );
	int maxDepth = 8;
	//the estimated complexity is O( N * log(N)^2 )
	while( true )
	{
		//distribute the points among the leafs and count their occupation
		distributeCells << < dim3( ( aPoints.size() + 31 ) / 32 ) , dim3( 32 , 1 , 1 ) >> > (
			thrust::raw_pointer_cast( dPoints.data() ) ,
			thrust::raw_pointer_cast( dPointsToLeafs.data() ) ,
			aPoints.size() ,
			rootX , rootY , rootSize ,
			thrust::raw_pointer_cast( dQuadNodes.data() )
			);
		copy( hQuadNodes , dQuadNodes );
		//split the fat leafs. terminate if there are no more fat leafs
		hSet< int > aLeafsIndices_c = aLeafsIndices;
		bool split = false;
		for( int leafIndex : aLeafsIndices_c )
		{
			if( hQuadNodes[ leafIndex ].itemsCount > 10 )
			{
				split = true;
				splitLeaf( hQuadNodes , leafIndex , aLeafsIndices );
			}
		}
		copy( dQuadNodes , hQuadNodes );
		if( !split || !--maxDepth )
		{
			break;
		}
	}
	//by now we have an empty balanced BVH ready to be filled in
	{
		dVector< int > dLeafsIndices , dLeafsScan;
		dLeafsIndices.resize( aLeafsIndices.size() );
		dLeafsScan.resize( aLeafsIndices.size() );
		thrust::copy( aLeafsIndices.begin() , aLeafsIndices.end() , dLeafsIndices.begin() );
		thrust::sort( dLeafsIndices.begin() , dLeafsIndices.end() );
		mapOrder << < dim3( ( aLeafsIndices.size() + 31 ) / 32 ) , dim3( 32 , 1 , 1 ) >> > (
			thrust::raw_pointer_cast( dQuadNodes.data() ) ,
			aLeafsIndices.size() ,
			thrust::raw_pointer_cast( dLeafsIndices.data() ) ,
			thrust::raw_pointer_cast( dLeafsScan.data() )
			);
		thrust::exclusive_scan( dLeafsScan.begin() , dLeafsScan.end() , dLeafsScan.begin() );
		fillLeafs << < dim3( ( aLeafsIndices.size() + 31 ) / 32 ) , dim3( 32 , 1 , 1 ) >> > (
			thrust::raw_pointer_cast( dQuadNodes.data() ) ,
			aLeafsIndices.size() ,
			thrust::raw_pointer_cast( dLeafsIndices.data() ) ,
			thrust::raw_pointer_cast( dLeafsScan.data() )
			);
		copy( hQuadNodes , dQuadNodes );
		/*hVector< int > hLeafsScan;
		hLeafsScan.resize( aLeafsIndices.size() );
		thrust::copy( dLeafsScan.begin() , dLeafsScan.end() , hLeafsScan.begin() );*/
	}


	//copy the tree to the host for debug usage
	out_aQuadNode.resize( dQuadNodes.size() );
	thrust::copy( dQuadNodes.begin() , dQuadNodes.end() , out_aQuadNode.begin() );
	//out_aQuadNode = dQuadNodes;

	//thrust::host_vector< int > counters = dCounters;
	/*thrust::device_vector< vec2 > dnPositions;
	dnPositions.reserve( positions.size() );
	kPack << < dim3( ( posN + 31 ) / 32 ) , dim3( 32 , 1 , 1 ) >> > (
		thrust::raw_pointer_cast< float* >( &dPositions[ 0 ] ) ,
		positions.size() ,
		thrust::raw_pointer_cast< float* >( &dnPositions[ 0 ] )
		);*/
}